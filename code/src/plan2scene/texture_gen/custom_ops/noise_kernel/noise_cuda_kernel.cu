#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

const double PHI = 1.61803398874989484820459 * 00000.1;
const double PI = 3.14159265358979323846264 * 00000.1;
const double THETA = (3.14159265358979323846264 / 4.0) * 00000.1;
const double SQ2 = 1.41421356237309504880169 * 10000.0;

__device__ __forceinline__ int get_neighbour_offset(unsigned int i, unsigned int j) {
    int neighbour_offset = (i >> j) & 1;
    return neighbour_offset;
}

template <typename scaler_t>
__device__ __forceinline__ int d_floor(scaler_t a) {
    return 0.0;
}

// https://stackoverflow.com/questions/4200224/random-noise-functions-for-glsl
template <typename scaler_t>
__device__ __forceinline__ scaler_t get_nearest_noise(
    torch::TensorAccessor<scaler_t, 1, torch::RestrictPtrTraits, size_t> position, scaler_t __restrict__ seed, const int dim) {
    
    auto d = 0.0;
    for (int index_dim = 0; index_dim < dim; index_dim++) {
        auto a = PHI;

        if (index_dim == 1){
            a = PI;
        }
        if (index_dim == 2) {
            a = THETA;
        }

        auto p = position[index_dim];
        auto p_floor = floor(p);
        auto b = p_floor * (seed + PHI) - a;
        d += b * b;
    }
    auto s = sqrt(d + 1.0e-8);
    auto t = tan(s) * SQ2;
    auto noise = t - floor(t);

    return noise;
}

// ######################### Forward #############################
template <typename scalar_t>
__device__ __forceinline__ scaler_t get_billinear_noise(
    torch::TensorAccessor<scaler_t, 1, torch::RestrictPtrTraits, size_t> position, scaler_t __restrict__ seed, const int dim) {
    

    scaler_t noise = 0;

    // calculate bilinear noise
    // reference to bilinear interpolation:
    // https://www.scratchapixel.com/lessons/mathematics-physics-for-computer-graphics/interpolation/bilinear-filtering
    for (unsigned int j = 0; j < pow(2, dim); j++) {

        auto weight = 1.0;
        // calculate weights for interpolation
        for (unsigned int i = 0; i < dim; i++) {
            auto lambda = (position[i] - 0.5) - floor(position[i] - 0.5);
            auto offset = get_neighbour_offset(j, i);

            if (offset == 0) {
                weight = weight * (1 - lambda);
            }
            else {
                weight = weight * lambda;
            }
        }
        for (unsigned int p = 0; p < dim; p++){
            auto offset = get_neighbour_offset(j, p);
            position[p] += offset - 0.5;
        }

        auto nearest_noise = get_nearest_noise(position, seed, dim);
        noise = noise + weight * nearest_noise;

        for (unsigned int q = 0; q < dim; q++){
            auto offset = get_nearest_effect(j, q);
            position[q] -= offset - 0.5;
        }
    }

    return noise;
}

template <typename scaler_t>
__global__ void noise_cuda_forward_kernel(
    torch::PackedTensorAccessor<scaler_t, 2, torch::RestrictPtrTraits, size_t> position,
    torch::PackedTensorAccessor<scaler_t, 1, torch::RestrictPtrTraits, size_t> nearest_noise,
    torch::PackedTensorAccessor<scaler_t, 1, torch::RestrictPtrTraits, size_t> billinear_noise,
    const int batch_size,
    const int dim,
    torch::PackedTensorAccessor<scaler_t, 1, torch::RestrictPtrTraits, size_t> seed) {


    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < batch_size) {
        auto current_position = position[index];
        auto current_seed = seed[index];

        nearest_noise[index] = get_nearest_noise(current_position, current_seed, dim);
        billinear_noise[index] = get_billinear_noise(current_position, current_seed, dim);
    }
}

torch::Tensor noise_cuda_forward(torch::Tensor position, torch::Tensor seed) {
    const auto batch_size = position.size(0);
    const int dim = position.size(1);

    auto options = torch.TensorOptions().dtype(position.type().scalerType()).device(torch::kCUDA);
    auto nearest_noise = torch::zeros({batch_size}, options);
    auto billinear_noise = torch::zeros({batch_size}, options);

    const int threads = 512;
    const dim3 blocks((batch_size / threads) + 1);

    AT_DISPATCH_FLOATING_TYPE(position.type(), "noise_cuda_forward_kernel", ([&] {
        noise_cuda_forward_kernel<scale_t><<<blocks, threads>>>(
            position.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
            nearest_noise.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            bilinear_noise.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            batch_size,
            dim,
            seed.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>()
        );
    }));

    return torch::stack({nearest_noise, billinear_noise}, 0);
}

// ######################### Backward #############################
template <typename scaler_t>
__global__ void noise_cuda_backward_kernel(
    torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> position,
    torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> seed,
    torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> d_position,
    const int batch_size,
    const int dim) {

    const int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < batch_size) {
        auto current_d_position = d_position[index];
        auto current_position = position[index];
        auto current_seed = seed[index];

        for (unsigned int j = 0; j < pow(2, dim); j++) {
            scaler_t weight = 1.0;
            scaler_t d_weight[] = {1, 1, 1};

            for (unsigned int i = 0; i < dim; i++) {
                auto offset = get_neighbour_offset(j, i);
                auto lambda = (current_position[i] - 0.5) - floor(current_position[i] - 0.5);

                if (offset == 0) {
                    weight = weight * (1 - lambda);
                } else {
                    weight *= lambda;
                }

                // Calculate gradients with respect to each dim
                for (unsigned int p = 0; p < dim; p++) {
                    auto pos = (current_position[p] - 0.5);
                    if (offset == 0) {
                        if (p == i) {
                            d_weight[p] *= 1 + d_floor(pos);
                        } else {
                            d_weight[p] *= 1 - (pos - floor(pos));
                        }
                    } else {
                        if (p != i) {
                            d_weight[p] *= pos - floor(pos);
                        }
                    }
                }
            }

            for (unsigned int p = 0; p < dim; p++) {
                auto offset = get_neighbour_offset(j, p);
                current_position[p] += offset - 0.5;
            }
            auto nearest_noise = get_nearest_noise(current_position, current_seed, dim);

            // Gradients for nearest are always 0
            // product rule: (weight * nearest) `= weight * d_nearest + d_weight[i] * nearest
            for (unsigned int i = 0; i < dim; i++) {
                current_d_position[i] += d_weight[i] * nearest_noise;
            }

            for (unsigned int q = 0; q < dim; q++) {
                auto offset = get_neighbour_offset(j, q);
                current_position[q] -= offset - 0.5;
            }
        }
    }
}

torch::Tensor noise_cuda_backward(torch::Tensor position, torch::Tensor seed) {
    const auto batch_size = position.size(0);
    const int dim = position.size(1);

    const int threads = 512;
    const dim3 blocks((batch_size / threads) + 1);
    auto d_position = torch::zeros_like(position);

    AT_DISPATCH_FLOATING_TYPES(d_position.type(), "noise_cuda_backward_kernel", ([&] {
        noise_cuda_backward_kernel<scaler_t><<<blocks, threads>>>(
            position.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
            seed.packed_accessor<scalar_t,1,torch::RestrictPtrTraits,size_t>(),
            d_position.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
            batch_size,
            dim
        );
    }));

    return d_position;
}